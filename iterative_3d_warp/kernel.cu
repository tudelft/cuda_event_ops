#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>


#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
        i += blockDim.x * gridDim.x)


__device__ bool check_out_of_bounds(
    float x, float y,
    float x_min, float x_max, float y_min, float y_max) {
    return (x < x_min || x >= x_max || y < y_min || y >= y_max);
}


__device__ void bilinear_interpolation(
    const float* flow_field,
    int height, int width, float x, float y,
    float& flow_x, float& flow_y) {

    int x0 = static_cast<int>(floor(x));
    int y0 = static_cast<int>(floor(y));
    int x1 = x0 + 1;
    int y1 = y0 + 1;

    float w00 = (x1 - x) * (y1 - y);  // top left
    float w01 = (x - x0) * (y1 - y);  // top right
    float w10 = (x1 - x) * (y - y0);  // bottom left
    float w11 = (x - x0) * (y - y0);  // bottom right

    flow_x = w00 * flow_field[(y0 * width + x0) * 2] +
             w01 * flow_field[(y0 * width + x1) * 2] +
             w10 * flow_field[(y1 * width + x0) * 2] +
             w11 * flow_field[(y1 * width + x1) * 2];

    flow_y = w00 * flow_field[(y0 * width + x0) * 2 + 1] +
             w01 * flow_field[(y0 * width + x1) * 2 + 1] +
             w10 * flow_field[(y1 * width + x0) * 2 + 1] +
             w11 * flow_field[(y1 * width + x1) * 2 + 1];
}


__global__ void iterative_3d_warp_kernel(
    const float* __restrict__ points, 
    const float* __restrict__ flow_fields, 
    float* __restrict__ warped_points,
    int batch_size, int num_points, int num_flow_fields, int num_z, int height, int width) {
    
    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx >= batch_size * num_points) return;

    // one thread can handle multiple points
    // useful when points >> threads
    // TODO: compare performance with one point per thread (above)
    CUDA_KERNEL_LOOP(idx, batch_size * num_points) {
        // indices
        int batch_idx = idx / num_points;
        int point_idx = idx % num_points;

        // load point coordinates
        float x = points[batch_idx * num_points * 4 + point_idx * 4];
        float y = points[batch_idx * num_points * 4 + point_idx * 4 + 1];
        float z = points[batch_idx * num_points * 4 + point_idx * 4 + 2];

        // keep track of original z value, value and out-of-bounds status
        float val = points[batch_idx * num_points * 4 + point_idx * 4 + 3];
        float z_orig = z;

        // if out of bounds to start with, return
        bool is_out_of_bounds = check_out_of_bounds(x, y, 0, width - 1, 0, height - 1);
        if (is_out_of_bounds) return;

        // warp forward: increasing z values
        // start with next integer z value
        int z_ceil = (z == ceil(z)) ? ceil(z) + 1 : ceil(z);
        for (int z1 = z_ceil; z1 < num_z; z1++) {
            int z0 = static_cast<int>(floor(z));
            float dz = z1 - z;

            // bilinear interpolation to get flow at (x, y)
            const float* flow_field = flow_fields + batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2;
            float flow_x, flow_y;
            bilinear_interpolation(flow_field, height, width, x, y, flow_x, flow_y);
            
            // warp point position
            // scale flow by dz
            x += flow_x * dz;
            y += flow_y * dz;
            z = z1;  // prevents rounding error?; same as z += dz

            // save warped point position
            int output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z1 * 5;
            warped_points[output_idx] = x;
            warped_points[output_idx + 1] = y;
            warped_points[output_idx + 2] = z;
            warped_points[output_idx + 3] = z_orig;
            warped_points[output_idx + 4] = val;

            // check bounds
            if (check_out_of_bounds(x, y, 0, width - 1, 0, height - 1)) {
                is_out_of_bounds = true;
                break;  // stop updating this point if it goes out of bounds
            }
        }

        // only do if not out of bounds
        if (!is_out_of_bounds) {
            // reload point coordinates
            x = points[batch_idx * num_points * 4 + point_idx * 4];
            y = points[batch_idx * num_points * 4 + point_idx * 4 + 1];
            z = points[batch_idx * num_points * 4 + point_idx * 4 + 2];

            // warp backward: decreasing z values
            // start with previous integer z value
            for (int z0 = static_cast<int>(floor(z)); z0 >= 0; z0--) {
                float dz = z - z0;

                // bilinear interpolation to get flow at (x, y)
                // flow always from z to z + 1 so use floor
                const float* flow_field = flow_fields + batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2;
                float flow_x, flow_y;
                bilinear_interpolation(flow_field, height, width, x, y, flow_x, flow_y);

                // warp point position
                // scale flow by dz
                x -= flow_x * dz;
                y -= flow_y * dz;
                z = z0;  // need int; same as z -= dz

                // save warped point position
                int output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z0 * 5;
                warped_points[output_idx] = x;
                warped_points[output_idx + 1] = y;
                warped_points[output_idx + 2] = z;
                warped_points[output_idx + 3] = z_orig;
                warped_points[output_idx + 4] = val;

                // check bounds
                if (check_out_of_bounds(x, y, 0, width - 1, 0, height - 1)) {
                    is_out_of_bounds = true;
                    break;  // stop updating this point if it goes out of bounds
                }
            }
        }

        // set all values to zero if out of bounds at some point
        if (is_out_of_bounds) {
            for (int z = 0; z < num_z; z++) {
                int output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z * 5;
                warped_points[output_idx + 4] = 0;
            }
        }
    }
}


__global__ void iterative_3d_warp_backward_kernel(
    const float* __restrict__ grad_output, 
    const float* __restrict__ points, 
    const float* __restrict__ flow_fields,
    const float* __restrict__ warped_points,
    float* __restrict__ grad_points,
    float* __restrict__ grad_flow_fields,
    int batch_size, int num_points, int num_flow_fields, int num_z, int height, int width) {

    // int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx >= batch_size * num_points) return;

    // one thread can handle multiple points
    // useful when points >> threads
    // TODO: compare performance with one point per thread (above)
    CUDA_KERNEL_LOOP(idx, batch_size * num_points) {
        // indices
        int batch_idx = idx / num_points;
        int point_idx = idx % num_points;

        // check if point was out of bounds: all values are zero
        if (warped_points[batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + 4] == 0) return;

        // load starting z
        float z_orig = points[batch_idx * num_points * 4 + point_idx * 4 + 2];

        // accumulate gradients for points
        float grad_warped_point_x = 0;
        float grad_warped_point_y = 0;

        // iterate over z values in reverse for forward warping gradient computation
        for (int z1 = num_z - 1; z1 > z_orig; z1--) {
            int z0 = z1 - 1;
            float dz = min(1.0f, z1 - z_orig);

            // get previous warped point position
            // final step: z0 is below or equal to z_orig
            // (in forward, we did z + 1 if z == ceil(z))
            float prev_x, prev_y;
            if (z0 <= z_orig) {
                prev_x = points[batch_idx * num_points * 4 + point_idx * 4];
                prev_y = points[batch_idx * num_points * 4 + point_idx * 4 + 1];
            } else {
                int prev_output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z0 * 5;
                prev_x = warped_points[prev_output_idx];
                prev_y = warped_points[prev_output_idx + 1];
            }

            // get bilinear interpolation weights
            int x0 = static_cast<int>(floor(prev_x));
            int y0 = static_cast<int>(floor(prev_y));
            int x1 = x0 + 1;
            int y1 = y0 + 1;

            float w00 = (x1 - prev_x) * (y1 - prev_y);  // top left
            float w01 = (prev_x - x0) * (y1 - prev_y);  // top right
            float w10 = (x1 - prev_x) * (prev_y - y0);  // bottom left
            float w11 = (prev_x - x0) * (prev_y - y0);  // bottom right

            // add output gradients
            int output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z1 * 5;
            grad_warped_point_x += grad_output[output_idx];
            grad_warped_point_y += grad_output[output_idx + 1];

            // add grads wrt x flow
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2], grad_warped_point_x * w00 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2], grad_warped_point_x * w01 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2], grad_warped_point_x * w10 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2], grad_warped_point_x * w11 * dz);
            
            // add grads wrt y flow
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2 + 1], grad_warped_point_y * w00 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2 + 1], grad_warped_point_y * w01 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2 + 1], grad_warped_point_y * w10 * dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2 + 1], grad_warped_point_y * w11 * dz);

            // calculate grad wrt xy
            // changes in flow field
            int idx_00 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2;
            int idx_01 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2;
            int idx_10 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2;
            int idx_11 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2;

            float f00_x = flow_fields[idx_00];
            float f01_x = flow_fields[idx_01];
            float f10_x = flow_fields[idx_10];
            float f11_x = flow_fields[idx_11];
            float f00_y = flow_fields[idx_00 + 1];
            float f01_y = flow_fields[idx_01 + 1];
            float f10_y = flow_fields[idx_10 + 1];
            float f11_y = flow_fields[idx_11 + 1];

            float dflowx_dx = (f01_x - f00_x) * (y1 - prev_y) + (f11_x - f10_x) * (prev_y - y0);
            float dflowy_dx = (f01_y - f00_y) * (y1 - prev_y) + (f11_y - f10_y) * (prev_y - y0);
            float dflowx_dy = (f10_x - f00_x) * (x1 - prev_x) + (f11_x - f01_x) * (prev_x - x0);
            float dflowy_dy = (f10_y - f00_y) * (x1 - prev_x) + (f11_y - f01_y) * (prev_x - x0);

            // add grads wrt x and y point
            // TODO: this looks wrong, but is correct?
            float grad_point_x = grad_warped_point_x * (1 + dflowx_dx * dz) + grad_warped_point_y * dflowy_dx * dz;
            float grad_point_y = grad_warped_point_x * dflowx_dy * dz + grad_warped_point_y * (1 + dflowy_dy * dz);
            grad_warped_point_x = grad_point_x;
            grad_warped_point_y = grad_point_y;
        }

        // reset gradients
        grad_warped_point_x = 0;
        grad_warped_point_y = 0;

        // iterate over z values in reverse for backward warping gradient computation
        for (int z0 = 0; z0 <= z_orig; z0++) {
            int z1 = z0 + 1;
            float dz = min(1.0f, z_orig - z0);

            // get previous warped point position
            // final step: z1 is larger than z_orig
            float prev_x, prev_y;
            if (z1 > z_orig) {
                prev_x = points[batch_idx * num_points * 4 + point_idx * 4];
                prev_y = points[batch_idx * num_points * 4 + point_idx * 4 + 1];
            } else {
                int prev_output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z1 * 5;
                prev_x = warped_points[prev_output_idx];
                prev_y = warped_points[prev_output_idx + 1];
            }

            // get bilinear interpolation weights
            // flow at previous position but left (z0) index
            int x0 = static_cast<int>(floor(prev_x));
            int y0 = static_cast<int>(floor(prev_y));
            int x1 = x0 + 1;
            int y1 = y0 + 1;

            float w00 = (x1 - prev_x) * (y1 - prev_y);  // top left
            float w01 = (prev_x - x0) * (y1 - prev_y);  // top right
            float w10 = (x1 - prev_x) * (prev_y - y0);  // bottom left
            float w11 = (prev_x - x0) * (prev_y - y0);  // bottom right

            // add output gradients
            int output_idx = batch_idx * num_points * num_z * 5 + point_idx * num_z * 5 + z0 * 5;
            grad_warped_point_x += grad_output[output_idx];
            grad_warped_point_y += grad_output[output_idx + 1];

            // add grads wrt x flow
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2], grad_warped_point_x * w00 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2], grad_warped_point_x * w01 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2], grad_warped_point_x * w10 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2], grad_warped_point_x * w11 * -dz);
            
            // add grads wrt y flow
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2 + 1], grad_warped_point_y * w00 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2 + 1], grad_warped_point_y * w01 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2 + 1], grad_warped_point_y * w10 * -dz);
            atomicAdd(&grad_flow_fields[batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2 + 1], grad_warped_point_y * w11 * -dz);

            // calculate grad wrt xy
            // changes in flow field
            int idx_00 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x0) * 2;
            int idx_01 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y0 * width + x1) * 2;
            int idx_10 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x0) * 2;
            int idx_11 = batch_idx * num_flow_fields * height * width * 2 + z0 * height * width * 2 + (y1 * width + x1) * 2;

            float f00_x = flow_fields[idx_00];
            float f01_x = flow_fields[idx_01];
            float f10_x = flow_fields[idx_10];
            float f11_x = flow_fields[idx_11];
            float f00_y = flow_fields[idx_00 + 1];
            float f01_y = flow_fields[idx_01 + 1];
            float f10_y = flow_fields[idx_10 + 1];
            float f11_y = flow_fields[idx_11 + 1];

            float dflowx_dx = (f01_x - f00_x) * (y1 - prev_y) + (f11_x - f10_x) * (prev_y - y0);
            float dflowy_dx = (f01_y - f00_y) * (y1 - prev_y) + (f11_y - f10_y) * (prev_y - y0);
            float dflowx_dy = (f10_x - f00_x) * (x1 - prev_x) + (f11_x - f01_x) * (prev_x - x0);
            float dflowy_dy = (f10_y - f00_y) * (x1 - prev_x) + (f11_y - f01_y) * (prev_x - x0);

            // add grads wrt x and y point
            // TODO: this looks wrong, but is correct?
            float grad_point_x = grad_warped_point_x * (1 + dflowx_dx * -dz) + grad_warped_point_y * dflowy_dx * -dz;
            float grad_point_y = grad_warped_point_x * dflowx_dy * -dz + grad_warped_point_y * (1 + dflowy_dy * -dz);
            grad_warped_point_x = grad_point_x;
            grad_warped_point_y = grad_point_y;
        }
    }
}


torch::Tensor iterative_3d_warp_cuda(
    torch::Tensor points,
    torch::Tensor flow_fields) {

    int batch_size = points.size(0);
    int num_points = points.size(1);
    int num_flow_fields = flow_fields.size(1);
    int num_z = num_flow_fields + 1;
    int height = flow_fields.size(2);
    int width = flow_fields.size(3);

    // points: (b, n, 4)
    // flow_fields: (b, d, h, w, 2)
    // warped_points: (b, n, d + 1, 5)
    auto warped_points = torch::zeros({batch_size, num_points, num_z, 5}, points.options());

    // one thread per point
    // TODO: less is more optimal?
    int threads = 1024;
    int blocks = (batch_size * num_points + threads - 1) / threads;

    iterative_3d_warp_kernel<<<blocks, threads>>>(
        points.data_ptr<float>(),
        flow_fields.data_ptr<float>(),
        warped_points.data_ptr<float>(),
        batch_size, num_points, num_flow_fields, num_z, height, width);

    return warped_points;
}


std::vector<torch::Tensor> iterative_3d_warp_backward_cuda(
    torch::Tensor grad_output,
    torch::Tensor points,
    torch::Tensor flow_fields,
    torch::Tensor warped_points) {

    int batch_size = points.size(0);
    int num_points = points.size(1);
    int num_flow_fields = flow_fields.size(1);
    int num_z = num_flow_fields + 1;
    int height = flow_fields.size(2);
    int width = flow_fields.size(3);

    auto grad_points = torch::zeros_like(points);
    auto grad_flow_fields = torch::zeros_like(flow_fields);

    // one thread per point
    // TODO: less is more optimal?
    int threads = 1024;
    int blocks = (batch_size * num_points + threads - 1) / threads;

    iterative_3d_warp_backward_kernel<<<blocks, threads>>>(
        grad_output.data_ptr<float>(),
        points.data_ptr<float>(),
        flow_fields.data_ptr<float>(),
        warped_points.data_ptr<float>(),
        grad_points.data_ptr<float>(),
        grad_flow_fields.data_ptr<float>(),
        batch_size, num_points, num_flow_fields, num_z, height, width);

    return {grad_points, grad_flow_fields};
}
